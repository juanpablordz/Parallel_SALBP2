#include "hip/hip_runtime.h"
//System Includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <iostream>
#include <vector>

//CUDA runtime 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Helper functions to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "hip/device_functions.h"

//Proyect files includes
#include "BinaryRestriction.h"
#include "VectorBR.h"
#include "BinaryMixer.h"

//Define Block SIZE.
int block_width = 1024;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: \"%s\": %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////PARALLEL MIX Function/////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void ParallelMix(unsigned long long int * d_cont ,unsigned int number_threads, int d_rest_size ,int numStations, unsigned int * d_temp, unsigned int * d_rest, unsigned int * d_out) {

	//Store thread ID
	const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	//Condition that verifies the usefulness of the thread.
	if (tid < number_threads) {

		//VARIABLES

		//This auxiliar variable signals when there has been a double activity assignment. 
		//IF (aux_v > 0) then a double assignment has been done. 
		unsigned int aux_v = 0;

		//This auxiliar variable stores information about which activities have been asigned.
		unsigned int cum_v = 0;

		//The purpose of flag is to identify when a double activity assignment has been made.
		bool flag = false;

		//This auxiliar vector has to be modified according to the number of stations in a given problem.
		unsigned int aux[7] = { 0,0,0,0,0,0,0 };

		//PRINTS
		//printf("Thread Num: %d \n", number_threads);
		//printf("Thread ID: %d \n", tid);

		//FOR-LOOP: Iterates over all solution tables from Constraint (Ri). 
		//Each thread (tid) is going to intersect every solution table in the new constraint (d_temp).
		for (int i = 0; i <d_rest_size ; i++) {

			//Perform OR operations on every line (STATION) of the solution table.

			//The objective is to identify if a single activity is  assigned to more than 1 station.
			for (int k = 0; k<numStations; k++) {

				//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
				////////////////////////////////////////////////BOOLEAN INTERSECTION ALGORITHM////////////////////////////////////////////////
				//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
				//1. Perform OR operation on table row (k) from tid solution table (d_temp) and new constraint (d_rest).
				//	 Store result in aux[k].cx
				//		1.1. Perform AND operation with aux[k] and cum_v. 
				//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

				aux[k] = *(d_temp + (tid*numStations + k)) | *(d_rest + k + (i*numStations));

				if (!flag) {
					aux_v = cum_v & aux[k];
					if (aux_v > 0) {
						flag = true;
					}
					cum_v = cum_v^aux[k];
				}
			}

			if (!flag) {
				//Tabla VALIDA
				aux_v = atomicAdd(d_cont, 1);

				for (int j = 0; j < numStations; j++) {
					*(d_out + (aux_v*numStations) + j) = aux[j];
					//printf("contador: %d\n", *d_cont);
				}
				//printf("VALIDA %d, %d\n", tid, i);
				flag = false;
				aux_v = 0;
				cum_v = 0;

			}
			else {
				//Tabla INVALIDA
				//printf("INVALIDA %d, %d\n", tid, i);
				flag = false;
				aux_v = 0;
				cum_v = 0;

			}
		}
		//Termina forloop
		//free memory from the variables

	}
}

extern "C" bool
runTest(int argc, const char **argv, unsigned long long int * h_cont, int numStations, unsigned int * h_temp, unsigned long long int h_temp_size , unsigned int * h_rest, int h_rest_size, unsigned int * h_out) {

	//use command-line specified CUDA device, otherwise use device with highest Gflops
	//es necesario?
	findCudaDevice(argc, (const char **)argv);

	//Number of threads and blocks
	unsigned int num_threads = h_temp_size / numStations;
	unsigned int num_blocks = 1 + (num_threads / block_width);
	//Prints
	cout << "Number of threads: " << num_threads << endl;
	cout << "Number of Blocks:  " << num_blocks << endl;
	printf("Out vector size: %s\n", to_string((h_temp_size/numStations) * (h_rest_size / numStations)));


	//GPU memory status variables
	size_t total_mem, free_mem;

	//Memory requirement for restriction
	const size_t mem_size_rest = sizeof(int) * h_rest_size;

	//Create Device Variables
	unsigned int * d_rest;
	unsigned int * d_out;
	unsigned long long int * d_cont;
	unsigned long long int * internal_cont = new unsigned long long int[1];
	*(internal_cont) = 0;


	//Allocate device memory CONT y REST
	checkCudaErrors(hipMalloc((void **)&d_rest, mem_size_rest));
	checkCudaErrors(hipMalloc((void **)&d_cont, sizeof(unsigned long long int)));

	//Copy memory from host to device CONT y REST
	checkCudaErrors(hipMemcpy(d_rest, h_rest, mem_size_rest, hipMemcpyHostToDevice));
	//realmente es necesario?
	gpuErrchk(hipMemcpy(d_cont, h_cont, sizeof(unsigned long long int), hipMemcpyHostToDevice));

	//GPU memory status REPORT.
	hipMemGetInfo(&free_mem, &total_mem);
	printf("Free Memory: %s \tTotal Memory: %s \n", to_string(free_mem), to_string(total_mem));


	//PRINT: Memory usage by vectors.
	/*
	cout << "Out vector byte size: " << to_string(mem_size_out) << endl;
	cout << "Rest vector size: " << h_rest_size << endl;
	
	*/

	//Declare a 1 MB constant
	const size_t MB = 1 << 20;

	//Buffer Conffiguration: It will manage the batches to be processed in the GPU
	hipError_t status;
	unsigned int * buffer;

	//Allocate at most 500MB of memory for the buffer size.
	//This limitation will allow us to allocate enough memory to store the resulting solutions vector.
	size_t buffer_size = std::min(4294967296 / (h_rest_size / numStations), (long long)536870912);
	//Make sure the allocation is succesful
	for (; buffer_size > MB; buffer_size -= MB) {
		//Allocate memory for buffer
		status = hipMalloc((void **)&buffer, buffer_size);
		if (status == hipSuccess)
			break;
	}

	//PRINT: Number of bytes allocated for GPU.
	cout << "Allocated " << buffer_size << " bytes on GPU. " << endl;

	//PRINT: Number of elements that must be processed.
	cout << h_temp_size << " int items require processing. " << endl;


	// Helps preserve the integrity of the solution vector
	// which has the size of the number of stations.
	size_t aux_buff_size = (buffer_size / sizeof(int)) % numStations;

	// Number of int elements that will be proccessed in this batch.
	// We substract the aux_buff_size in order to
	// garanty the integrity of the solution vectors.
	size_t batchN = (buffer_size - aux_buff_size) / sizeof(int);

	//The number fof elements that await processing.
	size_t remainN = h_temp_size;

	// Creation of output vector at most size Restriction.size() * Temporal.size()
	int elements_out = batchN * (h_rest_size / numStations);
	const size_t mem_size_out = elements_out * sizeof(int);

	//Allocate memory on GPU device
	checkCudaErrors(hipMalloc((void **)&d_out, mem_size_out));

	//PRINT: Batch size.
	cout << "Number of int proccessed in this batch: " << batchN << endl;

	//LOOP through host source data in batches.
	for (; remainN > 0; remainN -= batchN) {

		cout << "\n BATCH \n\n";

		//Si remain es menor que el tamano del batch,
		//ajusto el tamano del batch a lo que queda de datos
		//y en caso contrario lo dejo igual
		batchN = (remainN<batchN) ? remainN : batchN;
		size_t worksize = batchN * sizeof(int);
		cout << "Processing " << batchN << "/" << remainN << " items remaining." << endl;

		//COPY values from host temporal solution to buffer.
		//Process the items in buffer.
		hipMemcpy(buffer, h_temp, worksize, hipMemcpyHostToDevice);

		//The number of threads is determined by the number of Solution tables in the vector.
		num_threads = batchN / numStations;
		num_blocks = 1 + (num_threads / block_width);

		//Print: Number of Threads and Blocks to be used in this batch.
		cout << "Number of threads: " << num_threads << endl;
		cout << "Number of Blocks:  " << num_blocks << endl;

		/////////////////
		//Launch kernel//
		/////////////////
		ParallelMix << < num_blocks, block_width >> >(d_cont, num_threads, h_rest_size / numStations, numStations, buffer, d_rest, d_out);

		//Print: Report succesful execution.
		cout << "Succesful execution\n";

		//Copy BATCH counter from GPU to CPU. 
		gpuErrchk(hipMemcpy(h_cont, d_cont, sizeof(unsigned long long int), hipMemcpyDeviceToHost));

		//Acumulate value of BATCH COUNTERS IN A GLOBAL COUNTER.
		//BATCH COUNTER: h_cont and d_cont.
		//GLOBAL COUNTER: internal_cont
		*(internal_cont) += *(h_cont);

		//PRINT: Counters
		cout << *(h_cont) << endl;
		cout << "ACUM: " << *(internal_cont) << endl;

		//COPY valid solutions from device to host vector.
		gpuErrchk(hipMemcpy(h_out, d_out,*(h_cont)*numStations*sizeof(int), hipMemcpyDeviceToHost));
		
		///////////////////
		//UPDATE pointers//
		///////////////////

		//Increase Temp pointer in the magnitude of the proccessed batch.
		h_temp += batchN;
		//Increase the h_out pointer in the magnitude of the number of valid solutions from the proccessed batch.
		h_out = h_out + (*(h_cont)*numStations);

		//Reset. Value of out Batch counter and d_out vector space to be used.
		gpuErrchk(hipMemset(d_cont, (unsigned long long int)0, sizeof(unsigned long long int)));
		gpuErrchk(hipMemset(d_out, 0,mem_size_out));
	}

	//GPU memory status REPORT.
	hipMemGetInfo(&free_mem, &total_mem);
	printf("Free Memory: %s \tTotal Memory: %s \n", to_string(free_mem), to_string(total_mem));

	//Copy counter variable back to the host.
	*(h_cont) = *(internal_cont);

	//cleanup device memory
	//Reset Device
	hipDeviceReset();

	return 0;

}

////cleanup device memory
////checkCudaErrors(hipFree(d_out));
////checkCudaErrors(hipFree(d_temp));
////checkCudaErrors(hipFree(d_rest));
////checkCudaErrors(hipFree(buffer));

////Reset Device
//hipDeviceReset();

